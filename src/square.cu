#include <stdio.h>

#include <hip/hip_runtime.h>



hipError_t squareParallel(int* h_out, const int* h_in, size_t size); 

__global__ void squareKernel(int* d_out, int* d_in) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = d_in[idx];
	d_out[idx] = i * i;
}

hipError_t squareParallel(int* h_out, const int* h_in, size_t size) {
	hipError_t cudaStatus;

	int* d_out;
	int* d_in;

	cudaStatus = hipMalloc((void**)&d_out, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&d_in, size * sizeof(int));

	cudaStatus = hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);

	squareKernel<<<100, 450>>>(d_out, d_in);

	cudaStatus = hipGetLastError();

	cudaStatus = hipMemcpy(h_out, d_out, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);

	return cudaStatus;
}

/*
int main() {
	const size_t SIZE = 45000;
	int h_in[SIZE];
	for (size_t i = 0; i < SIZE; i++) {
		h_in[i] = i + 1;
	}
	int h_out[SIZE];

	squareParallel(h_out, h_in, SIZE);

	for (size_t i = 0; i < SIZE; i++) {
		printf("%d", h_out[i]);
		printf((i % 8) == 7 ? "\n" : "\t");
	}
	return 0;

}
*/
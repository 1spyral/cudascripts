#include <stdio.h>

#include <hip/hip_runtime.h>


#define MAX_PER_BLOCK 1024

inline __device__ int findBin(int min, int max, size_t bin_count, int val);

__global__ void histogramAtomicKernel(int* d_out, int* d_in, int min, int max, size_t bin_count, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        atomicAdd(&d_out[findBin(min, max, bin_count, d_in[idx])], 1);
    }
}

hipError_t histogramAtomicParallel(int* out, int* in, int min, int max, size_t bin_count, size_t size) {
    hipError_t cudaStatus;

    int* d_out;
    int* d_in;

    cudaStatus = hipMalloc(&d_out, bin_count * sizeof(int));
    cudaStatus = hipMalloc(&d_in, size * sizeof(int));

    cudaStatus = hipMemset(d_out, 0, bin_count * sizeof(int));
    cudaStatus = hipMemcpy(d_in, in, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDim((size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK);
    dim3 blockDim(gridDim.x > 1 ? MAX_PER_BLOCK : size);

    histogramAtomicKernel<<<gridDim, blockDim>>>(d_out, d_in, min, max, bin_count, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "histogramAtomicKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Exit;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Exit;
    }

    cudaStatus = hipMemcpy(out, d_out, bin_count * sizeof(int), hipMemcpyDeviceToHost);

Exit:
    hipFree(d_out);
    hipFree(d_in);
    return cudaStatus;
}

inline __device__ int findBin(int min, int max, size_t bin_count, int val) {
    return ((float)val - min) / ((float)max - min) * bin_count;
}

/*
int main() {
	int min = 5;
	int max = 15;
	int bin_count = 5;
	int size = 10;

	int in[] = { 6, 7, 8, 8, 9, 14, 10, 8, 5, 6 };
	int out[bin_count];
	printArray(in, size);
	histogramAtomicParallel(out, in, min, max, bin_count, size);
	printArray(out, bin_count);
}
*/
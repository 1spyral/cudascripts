#include <stdio.h>

#include <hip/hip_runtime.h>


// Note: if customizing MAX_PER_BLOCK, Blelloch only works with powers of 2 (binary tree)
#define MAX_PER_BLOCK 1024

hipError_t psaParallelHS(int* out, int* in, size_t size, bool inclusive = false, bool recursive = false);
hipError_t psaParallelB(int* out, int* in, size_t size, bool inclusive = false, bool recursive = false);

__global__ void psaHSKernel(int* d_out, int* d_in, int* d_offset, size_t size, bool inclusive) {
    __shared__ int psa[MAX_PER_BLOCK];

    size_t idx = threadIdx.x;
    size_t global_idx = blockIdx.x * MAX_PER_BLOCK + idx;

    size_t block_size = MAX_PER_BLOCK;
    if (blockIdx.x == (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
        block_size = size % MAX_PER_BLOCK;
        if (block_size == 0) {
            block_size = MAX_PER_BLOCK;
        }
    }

    size_t steps = ceil(log2f(block_size));
    size_t i = 1;
    size_t step = 1;

    if (steps == 0) {
        int val = d_in[global_idx];
        if (inclusive) {
            d_out[global_idx] = val;
            if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                d_offset[global_idx] = val;
            }
        } else {
            d_out[global_idx] = 0;
            if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                d_offset[global_idx] = val;
            }
        }
        return;
    }
    if (steps == 1) {
        int output = 0;
        if (idx < 2) {
            if (inclusive) {
                output += d_in[blockIdx.x * MAX_PER_BLOCK];
                if (idx == 1) {
                    output += d_in[global_idx];
                    if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                        d_offset[blockIdx.x] = output;
                    }
                }
            } else {
                if (idx == 1) {
                    output += d_in[blockIdx.x * MAX_PER_BLOCK];
                    if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                        d_offset[blockIdx.x] = output + d_in[global_idx];
                    }
                }
            }
        }
        __syncthreads();
        if (idx < 2) {
            d_out[global_idx] = output;
        }
        return;
    }

    if (idx < block_size) {
        if (idx == 0) {
            psa[0] = d_in[global_idx];
        } else {
            psa[idx] = d_in[global_idx - i] + d_in[global_idx];
        }
    }
    __syncthreads();

    for (i = 2; i < steps; i++) {
        int addend;
        step <<= 1;
        if (idx < block_size) {
            addend = idx < step ? 0 : psa[idx - step];
        }
        __syncthreads();
        if (idx < block_size) {
            psa[idx] += addend;
        }
        __syncthreads();
    }

    if (idx >= block_size) {
        return;
    }

    int output;

    step <<= 1;
    if (inclusive) {
        output = psa[idx] + (idx < step ? 0 : psa[idx - step]);
    } else {
        if (idx == 0) {
            output = 0;
        } else if (idx - 1 < step) {
            output = psa[idx - 1];
        } else {
            output = psa[idx - 1] + psa[idx - step - 1];
        }
    }

    d_out[global_idx] = output;
    if (idx == block_size - 1 && blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
        if (inclusive) {
            d_offset[blockIdx.x] = output;
        } else {
            d_offset[blockIdx.x] = psa[idx] + psa[idx - step];
        }
    }
}

__global__ void psaBKernel(int* d_out, int* d_in, int* d_offset, size_t size, bool inclusive) {
    __shared__ int psa[MAX_PER_BLOCK];

    size_t idx = threadIdx.x;
    size_t global_idx = blockIdx.x * MAX_PER_BLOCK + idx;

    size_t block_size = MAX_PER_BLOCK;
    if (blockIdx.x == (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
        block_size = size % MAX_PER_BLOCK;
        if (block_size == 0) {
            block_size = MAX_PER_BLOCK;
        }
    }

    size_t steps = ceil(log2f(block_size));
    size_t sweep_size = 1 << steps;
    size_t i = 1;
    size_t step = 2;
    size_t half_step = 1;

    if (steps == 0) {
        int val = d_in[global_idx];
        if (inclusive) {
            d_out[global_idx] = val;
            if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                d_offset[global_idx] = val;
            }
        } else {
            d_out[global_idx] = 0;
            if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                d_offset[global_idx] = val;
            }
        }
        return;
    }
    if (steps == 1) {
        int output = 0;
        if (idx < 2) {
            if (inclusive) {
                output += d_in[blockIdx.x * MAX_PER_BLOCK];
                if (idx == 1) {
                    output += d_in[global_idx];
                    if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                        d_offset[blockIdx.x] = output;
                    }
                }
            } else {
                if (idx == 1) {
                    output += d_in[blockIdx.x * MAX_PER_BLOCK];
                    if (blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
                        d_offset[blockIdx.x] = output + d_in[global_idx];
                    }
                }
            }
        }
        __syncthreads();
        if (idx < 2) {
            d_out[global_idx] = output;
        }
        return;
    }

    // Reduction
    if (idx < sweep_size) {
        int output = 0;
        if (idx < block_size) {
            output = d_in[global_idx];
        }
        if (idx & (step - 1) == half_step && idx < block_size) {
            output += d_in[global_idx - half_step];
        }
        psa[idx] = output;
    }
    __syncthreads();

    for (i = 2; i <= steps; i++) {
        int addend;
        step <<= 1;
        half_step <<= 1;
        if (idx < sweep_size) {
            addend = (((idx + 1) & (step - 1)) == 0) ? psa[idx - half_step] : 0;
        }
        __syncthreads();
        if (idx < sweep_size) {
            psa[idx] += addend;
        }
        __syncthreads();
    }

    // Downsweep
    if (idx == sweep_size - 1) {
        psa[idx] = 0;
    }
    __syncthreads();
    int output;
    for (i--; i > 1; i--) {
        if (idx < sweep_size) {
            if (((idx + 1) & (step - 1)) == 0) {
                output = psa[idx] + psa[idx - half_step];
            } else if (((idx + 1) & (step - 1)) == half_step) {
                output = psa[idx + half_step];
            } else {
                output = psa[idx];
            }
        }
        __syncthreads();
        if (idx < sweep_size) {
            psa[idx] = output;
        }
        __syncthreads();
        step >>= 1;
        half_step >>= 1;
    }

    if (idx < block_size) {
        if (inclusive) {
            if (idx == sweep_size - 1) {
                output = psa[idx - 1] + psa[idx] + d_in[global_idx];
            } else if (idx % 2 == 0) {
                output = psa[idx] + psa[idx + 1];
            } else {
                output = psa[idx + 2];
            }
        } else {
            if (((idx + 1) & (step - 1)) == 0) {
                output = psa[idx] + psa[idx - half_step];
            } else {
                output = psa[idx + half_step];
            }
        }
    }
    __syncthreads();
    if (idx < block_size) {
        d_out[global_idx] = output;
        if (idx == block_size - 1 && blockIdx.x != (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK - 1) {
            if (inclusive) {
                d_offset[blockIdx.x] = output;
            } else {
                d_offset[blockIdx.x] = output + d_in[global_idx];
            }
        }
    }
}

__global__ void addKernel(int* d_arr, int* d_in, int* d_offset, bool inclusive) {
    __shared__ int addend;

    int idx = threadIdx.x;
    int global_idx = (blockIdx.x + 1) * MAX_PER_BLOCK + idx;

    if (idx == 0) {
        addend = d_offset[blockIdx.x];
    }
    __syncthreads();

    d_arr[global_idx] += addend;
}

hipError_t psaParallelHS(int* out, int* in, size_t size, bool inclusive = false) {
    return psaParallelHS(out, in, size, inclusive, false);
}

hipError_t psaParallelHS(int* out, int* in, size_t size, bool inclusive, bool recursive) {
    hipError_t cudaStatus;

    int blocks = (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK;

    if (blocks == 0) {
        out = 0;
        goto Exit;
    }

    int* d_out;
    int* d_in;
    int* d_offset;

    if (recursive) {
        d_out = out;
        d_in = in;
    } else {
        cudaStatus = hipMalloc((void**) &d_out, size * sizeof(int));
        cudaStatus = hipMalloc((void**) &d_in, size * sizeof(int));
        
        cudaStatus = hipMemcpy(d_in, in, size * sizeof(int), hipMemcpyHostToDevice);
    }

    cudaStatus = hipMalloc((void**) &d_offset, (blocks - 1) * sizeof(int));
    
    psaHSKernel<<<blocks, MAX_PER_BLOCK>>>(d_out, d_in, d_offset, size, inclusive);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "psaHSKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Exit;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Exit;
    }

    if (blocks > 2) {
        cudaStatus = psaParallelHS(d_offset, d_offset, blocks - 1, true, true);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Recursive psaParallelHS failed: %s\n", hipGetErrorString(cudaStatus));
            goto Exit;
        }
    }

    if (blocks > 1) {
        addKernel<<<blocks - 1, MAX_PER_BLOCK>>>(d_out, d_in, d_offset, inclusive);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Exit;
        }
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Exit;
    }

    if (!recursive) {
        cudaStatus = hipMemcpy(out, d_out, size * sizeof(int), hipMemcpyDeviceToHost);
    }

Exit:
    if (!recursive) {
        hipFree(d_out);
        hipFree(d_in);
    }
    hipFree(d_offset);
    return cudaStatus;
}

hipError_t psaParallelB(int* out, int* in, size_t size, bool inclusive = false) {
    return psaParallelB(out, in, size, inclusive, false);
}

hipError_t psaParallelB(int* out, int* in, size_t size, bool inclusive, bool recursive) {
    hipError_t cudaStatus;

    int blocks = (size + MAX_PER_BLOCK - 1) / MAX_PER_BLOCK;

    if (blocks == 0) {
        out = 0;
        goto Exit;
    }

    int* d_out;
    int* d_in;
    int* d_offset;

    if (recursive) {
        d_out = out;
        d_in = in;
    } else {
        cudaStatus = hipMalloc((void**) &d_out, size * sizeof(int));
        cudaStatus = hipMalloc((void**) &d_in, size * sizeof(int));
        
        cudaStatus = hipMemcpy(d_in, in, size * sizeof(int), hipMemcpyHostToDevice);
    }

    cudaStatus = hipMalloc((void**) &d_offset, (blocks - 1) * sizeof(int));
    
    psaBKernel<<<blocks, MAX_PER_BLOCK>>>(d_out, d_in, d_offset, size, inclusive);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "psaBKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Exit;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Exit;
    }

    if (blocks > 2) {
        cudaStatus = psaParallelB(d_offset, d_offset, blocks - 1, true, true);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Recursive psaParallelB failed: %s\n", hipGetErrorString(cudaStatus));
            goto Exit;
        }
    }

    if (blocks > 1) {
        addKernel<<<blocks - 1, MAX_PER_BLOCK>>>(d_out, d_in, d_offset, inclusive);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Exit;
        }
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Exit;
    }

    if (!recursive) {
        cudaStatus = hipMemcpy(out, d_out, size * sizeof(int), hipMemcpyDeviceToHost);
    }
    
Exit:
    if (!recursive) {
        hipFree(d_out);
        hipFree(d_in);
    }
    hipFree(d_offset);
    return cudaStatus;
}

/*
int main() {
	int size = 10;
	int in[size];
	int out[size];
	for (int i = 1; i <= size; i++) {
		in[i - 1] = i;
	}
	printArray(in, size);
	psaParallelHS(out, in, size, false);
	printArray(out, size);
}
*/
/*
int main() {
	int size = 30;
	int in[size];
	int out[size];
	for (int i = 1; i <= size; i++) {
		in[i - 1] = i;
	}
	printArray(in, size);
	psaParallelB(out, in, size, false);
	printArray(out, size);
	psaParallelB(out, in, size, true);
	printArray(out, size);
}
*/
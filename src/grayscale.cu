#include "hip/hip_runtime.h"
// Conversion reference:
// https://en.wikipedia.org/wiki/Grayscale#Converting_color_to_grayscale

#include <stdio.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <>

#include <opencv2/core.hpp>

#include "images.h"

__device__ float linearize(float gamma);
__device__ float convertToGamma(float linear);
__device__ inline float scale1(uint8_t x);
__device__ inline uint8_t scale255(float x);

__global__ void grayscaleKernel(uint8_t* d_out, uint8_t* d_in, size_t size) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= size) {
        return;
    }
    float b = scale1(d_in[idx * 3]);
    float g = scale1(d_in[idx * 3 + 1]);
    float r = scale1(d_in[idx * 3 + 2]);

    float linearR = linearize(r);
    float linearG = linearize(g);
    float linearB = linearize(b);

    uint8_t intensity = scale255(convertToGamma(linearR * 0.2126 + linearG * 0.7152 + 0.0722 * linearB));

    d_out[idx] = intensity;
}

hipError_t grayscaleParallel(cv::Mat &h_out, cv::Mat &h_in) {
    hipError_t cudaStatus;

    size_t size = sizeInPixels(h_in);

    uint8_t* h_arr_in = flattenColor(h_in);
    uint8_t* h_arr_out = new uint8_t[size];

    uint8_t* d_arr_in;
    uint8_t* d_arr_out;

    int threadsPerBlock = 1024;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    cudaStatus = hipMalloc((void**)&d_arr_in, size * 3);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_arr_in!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_arr_out, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for d_arr_out!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_arr_in, h_arr_in, size * 3, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for d_arr_in!");
        goto Error;
    }

    grayscaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_arr_out, d_arr_in, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "grayscaleKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching grayscaleKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(h_arr_out, d_arr_out, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for h_arr_out!");
        goto Error;
    }

    // Copy the output array back to the cv::Mat
    h_out = cv::Mat(h_in.rows, h_in.cols, CV_8UC1, h_arr_out);

Error:
    hipFree(d_arr_in);
    hipFree(d_arr_out);
    return cudaStatus;
}

__device__ float linearize(float gamma) {
    if (gamma <= 0.04045) {
        return gamma / 12.92;
    }
    return pow((gamma + 0.055) / 1.055, 2.4);
}

__device__ float convertToGamma(float linear) {
    if (linear <= 0.0031308) {
        return 12.92 * linear;
    }
    return 1.055 * pow(linear, 1 / 2.4) - 0.055;
}

__device__ inline float scale1(uint8_t x) {
    return x / 255.0;
}

__device__ inline uint8_t scale255(float x) {
    return (uint8_t)(x * 255);
}

/*
int main() {
	std::string PATH = "images/starrynight.jpg";
	cv::Mat img = getImage(PATH);
	previewImage(img, 500, 500);
	grayscaleParallel(img, img);
	previewImage(img, 500, 500);
}
*/